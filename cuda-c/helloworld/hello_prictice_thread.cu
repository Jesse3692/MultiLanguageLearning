
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void helloFromGPU(void){
    printf("Hello World from GPU thread %d!\n", threadIdx.x);
}

int main(void){
    // hello from cpu
    printf("Hello World from CPU!\n");

    // hello from gpu
    helloFromGPU<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}