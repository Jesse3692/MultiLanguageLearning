#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mathKernel1(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float a, b;
  a = b = 0.0f;

  if (tid % 2 == 0) {
    a = 100.0f;
  } else {
    b = 200.0f;
  }
  c[tid] = a + b;
}

__global__ void mathKernel2(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float a, b;
  a = b = 0.0f;
  if ((tid / warpSize) % 2 == 0) {
    a = 100.0f;
  } else {
    b = 200.0f;
  }
  c[tid] = a + b;
}

__global__ void mathKernel3(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  bool ipred = (tid % 2 == 0);
  if (ipred) {
    ia = 100.0f;
  }
  if (!ipred) {
    ib = 200.0f;
  }
  c[tid] = ia + ib;
}

__global__ void mathKernel4(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  int itid = tid >> 5;
  if (itid & 0x01 == 0) {
    ia = 100.0f;
  } else {
    ib = 200.0f;
  }
  c[tid] = ia + ib;
}

__global__ void warmingup(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  if ((tid / warpSize) % 2 == 0) {
    ia = 100.0f;
  } else {
    ib = 200.0f;
  }
  c[tid] = ia + ib;
}

int main(int argc, char **argv) {
  // set up device
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

  // set up data size
  int size = 64;
  int blocksize = 64;
  if (argc > 1)
    blocksize = atoi(argv[1]);
  if (argc > 2)
    blocksize = atoi(argv[2]);
  printf("Data size %d ", size);

  // set up execution configuration
  dim3 block(blocksize, 1);
  dim3 grid((size + block.x - 1) / block.x, 1);
  printf("Execution Configure (block %dgrid %d)\n", block.x, grid.x);

  // allocate gpu memory
  float *d_C;
  size_t nBytes = size * sizeof(float);
  hipMalloc((float **)&d_C, nBytes);

  // run a warmup kernel to remove overhead
  size_t iStart, iElaps;
  hipDeviceSynchronize();
  iStart = cpuSecond() - iStart;
  warmingup<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  iElaps = cpuSecond() - iStart;
  printf("warmup <<< %4d %4d >>> elapsed %d sec \n", grid.x, block.x, iElaps);

  // run kernel 1
  iStart = cpuSecond();
  mathKernel1<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  iElaps = cpuSecond() - iStart;
  printf("mathKernel1 <<< %4d %4d >>> elapsed %d sec \n", grid.x, block.x,
         iElaps);

  // run kernel 2
  iStart = cpuSecond();
  mathKernel2<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  iElaps = cpuSecond() - iStart;
  printf("mathKernel2 <<< %4d %4d >>> elapsed %d sec \n", grid.x, block.x,
         iElaps);

  // run kernel 3
  iStart = cpuSecond();
  mathKernel3<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  iElaps = cpuSecond() - iStart;
  printf("mathKernel3 <<< %4d %4d >>> elapsed %d sec \n", grid.x, block.x,
         iElaps);

  // run kernel 4
  iStart = cpuSecond();
  mathKernel4<<<grid, block>>>(d_C);
  hipDeviceSynchronize();
  iElaps = cpuSecond() - iStart;
  printf("mathKernel4 <<< %4d %4d >>> elapsed %d sec \n", grid.x, block.x,
         iElaps);

  // free gpu memory and reset divece
  hipFree(d_C);
  hipDeviceReset();
  return EXIT_SUCCESS;
}
